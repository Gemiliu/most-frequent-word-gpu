#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 * Modifications to run Trie Min Heap done by Jeremy Villalobos (2015)
 * https://www.elance.com/s/jfvillal/
 */
#include <stdio.h>
#include <stdlib.h>

#include <string.h>
#include <ctype.h>

# define MAX_CHARS 26
# define MAX_WORD_SIZE 30
#include<stdbool.h>

// A Trie node
typedef struct TrieNode TrieNode;
struct TrieNode {
	bool isEnd; // indicates end of word
	unsigned frequency;  // the number of occurrences of a word
	int indexMinHeap; // the index of the word in minHeap
	TrieNode* child[MAX_CHARS]; // represents 26 slots each for 'a' to 'z'.
};
//typedef struct TrieNode TrieNode;

// A Min Heap node
struct MinHeapNode {
	TrieNode* root; // indicates the leaf node of TRIE
	unsigned frequency; //  number of occurrences
	char* word; // the actual word stored
};
typedef struct MinHeapNode MinHeapNode;
// A Min Heap
typedef struct MinHeap MinHeap;
struct MinHeap {
	unsigned capacity; // the total size a min heap
	int count; // indicates the number of slots filled.
	MinHeapNode* array; //  represents the collection of minHeapNodes
};

//void printKMostFreq(FILE* fp, int k);
__device__ size_t mystrlen(const char* str);
__device__ char* mystrcpy(char *s1, const char *s2);
//__device__ char getLetter(int id);
__device__ void displayMinHeap(MinHeap* minHeap, char* ans, int buff_size,
		unsigned* count, int thread_id, int* lineup, int* win, char* top_words,
		unsigned int* top_words_count);
__device__ void insertTrieAndHeap(const char *word, TrieNode** root,
		MinHeap* minHeap);
__device__ void insertUtil(TrieNode** root, MinHeap* minHeap, const char* word,
		const char* dupWord);
__device__ void insertUtil(TrieNode** root, MinHeap* minHeap, const char* word,
		const char* dupWord);
__device__ void insertInMinHeap(MinHeap* minHeap, TrieNode** root,
		const char* word);
__device__ void buildMinHeap(MinHeap* minHeap);
__device__ void minHeapify(MinHeap* minHeap, int idx);
__device__ void swapMinHeapNodes(MinHeapNode* a, MinHeapNode* b);
__device__ MinHeap* createMinHeap(int capacity);
__device__ TrieNode* newTrieNode();

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

#define WORKERS 12
#define BUFF_SIZE 16
#define K_WORDS 5

__global__ void k_words(char * data, int size, char* ans, unsigned * counts,
		int* lineup, int* win, char* top_words, unsigned int* top_words_count) {

//	if (threadIdx.x == 0) {
	// Create a Min Heap of Size k
	MinHeap* minHeap = createMinHeap(K_WORDS);

	// Create an empty Trie
	TrieNode* root = NULL;

	// A buffer to store one word at a time
	char buffer[BUFF_SIZE];

	// Read words one by one from file.  Insert the word in Trie and Min Heap
	int m = 0;
	for (int i = 0; i < size; i++) {
		if (data[i] == ' ') {
			if (m > 1) {
				//process word
				buffer[m] = '\0';
				/***
				 * The letters are distributed among the processors.
				 * There can only be up to 28 workers for this implementation
				 */
				if (threadIdx.x == (buffer[0] - 97) % WORKERS) {
					insertTrieAndHeap(buffer, &root, minHeap);
				}
			}
			m = 0;
		} else {
			buffer[m] = data[i];
			++m;
			/**
			 * ignore words longer than  BUFF_SIZE bytes
			 */
			if (m >= BUFF_SIZE - 1) {
				m = 0;
			}
		}

//	    	while( fscanf( fp, "%s", buffer ) != EOF )
//	    		insertTrieAndHeap(buffer, &root, minHeap);

		// The Min Heap will have the k most frequent words, so print Min Heap nodes

	}

	displayMinHeap(minHeap, ans, BUFF_SIZE, counts, threadIdx.x, lineup, win,
			top_words, top_words_count);

//	}
}

//__device__ char getLetter(int id) {
//	return id + 97;
//}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(int argc, char** argv) {
	char *d = NULL;
	char *ans = NULL;
	unsigned *counts = NULL;
	int *lineup = NULL;
	int *win = NULL;
	char* top_words;
	unsigned int* top_words_count;

	char *host_ans = new char[K_WORDS * BUFF_SIZE * WORKERS];
	unsigned *host_counts = new unsigned[K_WORDS * WORKERS];

	char * host_top_words = new char[K_WORDS * BUFF_SIZE];
	unsigned *host_top_words_count = new unsigned[K_WORDS];

	int *host_win = new int[K_WORDS];

	clock_t begin, end;
	double time_spent;

	begin = clock();

	FILE *fp =
			fopen(	argv[1],
					"r");
	if (fp == NULL)
		printf("File doesn't exist ");
	else {

		fseek(fp, 0, SEEK_END);
		long fsize = ftell(fp);
		fseek(fp, 0, SEEK_SET);

		char *string = (char*) malloc(fsize + 1);
		fread(string, fsize, 1, fp);
		fclose(fp);

		string[fsize] = 0;

//		printf("file characters:\n%s\n", string);

		CUDA_CHECK_RETURN(hipMalloc((char** ) &d, sizeof(char) * fsize));

		CUDA_CHECK_RETURN(
				hipMalloc((char** ) &ans, sizeof(char) * K_WORDS * BUFF_SIZE * WORKERS));

		CUDA_CHECK_RETURN(
				hipMalloc((int** ) &counts, sizeof(unsigned) * K_WORDS * WORKERS));

		CUDA_CHECK_RETURN(hipMalloc((int** ) &lineup, sizeof(int) * WORKERS));

		CUDA_CHECK_RETURN(hipMalloc((int** ) &win, sizeof(int) * K_WORDS));

		CUDA_CHECK_RETURN(
				hipMalloc((char** ) &top_words, sizeof(char) * K_WORDS * BUFF_SIZE));

		CUDA_CHECK_RETURN(
				hipMalloc((unsigned int** ) &top_words_count, sizeof(int) * K_WORDS ));

		CUDA_CHECK_RETURN(
				hipMemcpy(d, string, sizeof(char) * fsize,
						hipMemcpyHostToDevice));

		for (int m = 0; m < WORKERS * K_WORDS * BUFF_SIZE; m++) {
			host_ans[m] = ' ';
		}
		for (int m = 0; m < WORKERS * K_WORDS; m++) {
			host_counts[m] = 0;
		}
		CUDA_CHECK_RETURN(
				hipMemcpy(ans, host_ans, sizeof(char) * K_WORDS *BUFF_SIZE, hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(
				hipMemcpy(counts, host_counts, sizeof(unsigned) * K_WORDS , hipMemcpyHostToDevice));
//		printf("Size %d \n");
		k_words<<<1, WORKERS>>>(d, fsize, ans, counts, lineup, win, top_words,
				top_words_count);

		CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
		CUDA_CHECK_RETURN(hipGetLastError());
		CUDA_CHECK_RETURN(
				hipMemcpy(string, d, sizeof(char) * fsize,
						hipMemcpyDeviceToHost));

		CUDA_CHECK_RETURN(
				hipMemcpy(host_ans, ans, sizeof(char) * K_WORDS * BUFF_SIZE * WORKERS, hipMemcpyDeviceToHost));

		CUDA_CHECK_RETURN(
				hipMemcpy(host_counts, counts, sizeof(unsigned) * K_WORDS * WORKERS, hipMemcpyDeviceToHost));

		CUDA_CHECK_RETURN(
				hipMemcpy(host_win,win, sizeof(int) * K_WORDS, hipMemcpyDeviceToHost));

		CUDA_CHECK_RETURN(
				hipMemcpy(host_top_words, top_words, sizeof(char) * K_WORDS * BUFF_SIZE, hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(
				hipMemcpy(host_top_words_count, top_words_count, sizeof(int) * K_WORDS, hipMemcpyDeviceToHost));

//		for (int m = 0; m < WORKERS; m++) {
//			printf("\nworker: %d  \n", m);
//			for (int i = 0; i < K_WORDS; i++) {
//				printf("\n%u ", host_counts[m * K_WORDS + i]);
//				for (int j = 0; j < BUFF_SIZE; j++) {
//					printf("%c",
//							host_ans[m * (BUFF_SIZE * K_WORDS) + (i * BUFF_SIZE)
//									+ j]);
//				}
//				printf("");
//			}
//		}

		printf("\n***Result***\n");

		int k_words = 0;
		for (int k = 0; k < K_WORDS; k++) {
			unsigned max = 0;

			for (int m = 0; m < WORKERS; m++) {
				for (int i = 0; i < K_WORDS; i++) {
					if (max < host_counts[m * K_WORDS + i]) {
						max = host_counts[m * K_WORDS + i];
					}
				}
			}

			for (int m = 0; m < WORKERS; m++) {
				for (int i = 0; i < K_WORDS; i++) {
					if (max == host_counts[m * K_WORDS + i]) {
						printf("%u word: ", host_counts[m * K_WORDS + i]);
						host_counts[m * K_WORDS + i] = 0;
						for (int j = 0; j < BUFF_SIZE; j++) {
							printf("%c",
									host_ans[m * (BUFF_SIZE * K_WORDS)
											+ i * BUFF_SIZE + j]);
						}
						printf("\n");
						++k_words;
					}
					if (k_words == K_WORDS) {
						break;
					}

				}
				if (k_words == K_WORDS) {
					break;
				}
			}
			if (k_words == K_WORDS) {
				break;
			}
		}

//		for (int i = 0; i < K_WORDS; i++) {
//			printf("\ni %d win %d ", i, host_win[i]);
//		}

		CUDA_CHECK_RETURN(hipFree((void* ) d));
		CUDA_CHECK_RETURN(hipDeviceReset());

		/* here, do your time-consuming job */
		end = clock();
		time_spent = (double) (end - begin) ;/// CLOCKS_PER_SEC;

		printf("\nTime: %f \n", time_spent);
	}
	return 0;
}

// A utility function to create a new Trie node
__device__ TrieNode* newTrieNode() {
	// Allocate memory for Trie Node
	TrieNode* trieNode = (TrieNode *) malloc(sizeof(TrieNode));

	// Initialize values for new node
	trieNode->isEnd = 0;
	trieNode->frequency = 0;
	trieNode->indexMinHeap = -1;
	int i;
	for (i = 0; i < MAX_CHARS; ++i)
		trieNode->child[i] = NULL;

	return trieNode;
}

// A utility function to create a Min Heap of given capacity
__device__ MinHeap* createMinHeap(int capacity) {
	MinHeap* minHeap = (MinHeap *) malloc(sizeof(MinHeap*));

	minHeap->capacity = capacity;
	minHeap->count = 0;

	// Allocate memory for array of min heap nodes
	minHeap->array = (MinHeapNode *) malloc(sizeof(MinHeapNode) * capacity);//new MinHeapNode [ minHeap->capacity ];

	return minHeap;
}

// A utility function to swap two min heap nodes. This function
// is needed in minHeapify
__device__ void swapMinHeapNodes(MinHeapNode* a, MinHeapNode* b) {
	MinHeapNode temp = *a;
	*a = *b;
	*b = temp;
}

// This is the standard minHeapify function. It does one thing extra.
// It updates the minHapIndex in Trie when two nodes are swapped in
// in min heap
__device__ void minHeapify(MinHeap* minHeap, int idx) {
	int left, right, smallest;

	left = 2 * idx + 1;
	right = 2 * idx + 2;
	smallest = idx;
	if (left < minHeap->count
			&& minHeap->array[left].frequency
					< minHeap->array[smallest].frequency)
		smallest = left;

	if (right < minHeap->count
			&& minHeap->array[right].frequency
					< minHeap->array[smallest].frequency)
		smallest = right;

	if (smallest != idx) {
		// Update the corresponding index in Trie node.
		minHeap->array[smallest].root->indexMinHeap = idx;
		minHeap->array[idx].root->indexMinHeap = smallest;

		// Swap nodes in min heap
		swapMinHeapNodes(&minHeap->array[smallest], &minHeap->array[idx]);

		minHeapify(minHeap, smallest);
	}
}

// A standard function to build a heap
__device__ void buildMinHeap(MinHeap* minHeap) {
	int n, i;
	n = minHeap->count - 1;

	for (i = (n - 1) / 2; i >= 0; --i)
		minHeapify(minHeap, i);
}

// Inserts a word to heap, the function handles the 3 cases explained above
__device__ void insertInMinHeap(MinHeap* minHeap, TrieNode** root,
		const char* word) {
	// Case 1: the word is already present in minHeap
	if ((*root)->indexMinHeap != -1) {
		++(minHeap->array[(*root)->indexMinHeap].frequency);

		// percolate down
		minHeapify(minHeap, (*root)->indexMinHeap);
	}

	// Case 2: Word is not present and heap is not full
	else if (minHeap->count < minHeap->capacity) {
		int count = minHeap->count;
		minHeap->array[count].frequency = (*root)->frequency;
		minHeap->array[count].word = (char *) malloc(
				sizeof(char) * mystrlen(word) + 1);
		mystrcpy(minHeap->array[count].word, word);

		minHeap->array[count].root = *root;
		(*root)->indexMinHeap = minHeap->count;

		++(minHeap->count);
		buildMinHeap(minHeap);
	}

	// Case 3: Word is not present and heap is full. And frequency of word
	// is more than root. The root is the least frequent word in heap,
	// replace root with new word
	else if ((*root)->frequency > minHeap->array[0].frequency) {

		minHeap->array[0].root->indexMinHeap = -1;
		minHeap->array[0].root = *root;
		minHeap->array[0].root->indexMinHeap = 0;
		minHeap->array[0].frequency = (*root)->frequency;

		// delete previously allocated memoory and
		free(minHeap->array[0].word);
		minHeap->array[0].word = (char *) malloc(
				sizeof(char) * mystrlen(word) + 1);
		mystrcpy(minHeap->array[0].word, word);

		minHeapify(minHeap, 0);
	}
}

__device__ size_t mystrlen(const char *str) {
	register const char *s;

	for (s = str; *s; ++s)
		;
	return (s - str);
}

__device__ char* mystrcpy(char *s1, const char *s2) {
	char *s = s1;
	while ((*s++ = *s2++) != 0)
		;
	return (s1);
}

// Inserts a new word to both Trie and Heap
__device__ void insertUtil(TrieNode** root, MinHeap* minHeap, const char* word,
		const char* dupWord) {
	// Base Case
	if (*root == NULL)
		*root = newTrieNode();

	//  There are still more characters in word
	if (*word != '\0')
		insertUtil(&((*root)->child[*word - 97]), minHeap, word + 1, dupWord);
	else // The complete word is processed
	{
		// word is already present, increase the frequency
		if ((*root)->isEnd)
			++((*root)->frequency);
		else {
			(*root)->isEnd = 1;
			(*root)->frequency = 1;
		}

		// Insert in min heap also
		insertInMinHeap(minHeap, root, dupWord);
	}
}

// add a word to Trie & min heap.  A wrapper over the insertUtil
__device__ void insertTrieAndHeap(const char *word, TrieNode** root,
		MinHeap* minHeap) {
	insertUtil(root, minHeap, word, word);
}

// A utility function to show results, The min heap
// contains k most frequent words so far, at any time
//
__device__ void displayMinHeap(MinHeap* minHeap, char* ans, int buff_size,
		unsigned* count, const int thread_id, int* lineup, int* win,
		char* top_words, unsigned int* top_words_count) {

	int i;

	for (i = 0; i < minHeap->count; ++i) {
		for (int j = 0; j < buff_size; j++) {
			ans[(thread_id * K_WORDS * BUFF_SIZE) + (i * buff_size) + j] =
					minHeap->array[i].word[j];
		}
		count[thread_id * K_WORDS + i] = minHeap->array[i].frequency;
		//		printf("%s : %d\n", minHeap->array[i].word,
		//				minHeap->array[i].frequency);
	}

	for (i = minHeap->count; i < K_WORDS; ++i) {
		ans[(thread_id * K_WORDS * BUFF_SIZE) + (i * buff_size)] = '\0';
		count[thread_id * K_WORDS + i] = 0;
	}

}

